#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include "itap.hpp"

#define BLOCK_SIZE 512 

namespace itap {

void checkError_t(hipError_t error, std::string msg) {
    if (error != hipSuccess) {
        printf("%s: %d\n", msg.c_str(), error);
        std::exit(1);
    }
}

__global__ void partition_gpu(
  int* d_adjp, int* d_adjncy, int* d_adjncy_size, int* d_dep_size,
  int* d_topo_result_gpu,
  int* d_partition_result_gpu,
  int* d_partition_counter_gpu,
  int partition_size,
  int* max_partition_id,
  int read_offset, uint32_t read_size, // [read_offset, read_offset + read_size - 1] are all the frontiers 
  uint32_t* write_size,
  int* d_fu_partition // the future partition this node will be assigned to(if partition not full)
) {

  uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;

  if(tid < read_size) {
    int cur_id = d_topo_result_gpu[read_offset + tid]; // get current task id
    if(d_adjp[cur_id] == -1) {
      return; // if _adjp[cur_id] = -1, that means it has no fanout
    }
    for(int offset=d_adjp[cur_id]; offset<d_adjp[cur_id] + d_adjncy_size[cur_id]; offset++) {
      int neighbor_id = d_adjncy[offset];
      
      atomicMax(&d_fu_partition[neighbor_id], d_partition_result_gpu[cur_id]);

      if(atomicSub(&d_dep_size[neighbor_id], 1) == 1) {
        int position = atomicAdd(write_size, 1); // no need to atomic here...
        d_topo_result_gpu[read_offset + read_size + position] = neighbor_id;        
        int cur_partition = d_fu_partition[neighbor_id]; // get the largest partition id from the parent  
        if(atomicAdd(&d_partition_counter_gpu[cur_partition], 1) < partition_size) { 
          d_partition_result_gpu[neighbor_id] = cur_partition; // no need to atomic here cuz only one thread will access this neighbor here
        }
        else {
          int new_partition_id = atomicAdd(max_partition_id, 1) + 1; // now we have new partition when we find cur_partition is full
                                                                     // we need to store this new partition id locally to the thread
          d_partition_result_gpu[neighbor_id] = new_partition_id;
          d_partition_counter_gpu[new_partition_id]++;  
        }
      }
    }
  }
}



void iTAP::_partition_cuda() {

  _export_csr();   

  std::vector<int> source;
  for(const auto& node : _nodes) {
    if(node._fanins.size() == 0) {
      source.push_back(node._csr_id);
    }
  }

  int* d_adjp; 
  int* d_adjncy; 
  int* d_adjncy_size;
  int* d_dep_size;
  int* d_topo_result_gpu;
  int* d_partition_result_gpu;
  int* d_partition_counter_gpu;
  int read_offset = 0;
  uint32_t read_size = source.size();
  uint32_t* write_size;
  int* max_partition_id; // max_partition id we have currently, initially is source.size() - 1

  unsigned num_nodes = _adjp.size();
  unsigned num_edges = _adjncy.size();
  
  std::vector<int> fu_partition(num_nodes, -1);
  int* d_fu_partition;


  std::vector<int> topo_result_gpu(num_nodes);
  std::vector<int> partition_result_gpu(num_nodes, -1);
  int source_partition_id = 0;
  for(unsigned i=0; i<source.size(); i++) {
    partition_result_gpu[source[i]] = source_partition_id;
    source_partition_id++;
  }
  std::vector<int> partition_counter_gpu(num_nodes, 0);
  for(unsigned i=0; i<source.size(); i++) { // at the beginning, each source corresponds to one partition
    partition_counter_gpu[i]++;
  }

  checkError_t(hipMalloc(&d_adjp, sizeof(int)*num_nodes), "d_adjp allocation failed");
  checkError_t(hipMalloc(&d_adjncy, sizeof(int)*num_edges), "d_adjncy allocation failed");
  checkError_t(hipMalloc(&d_adjncy_size, sizeof(int)*num_nodes), "d_adjncy_size allocation failed");
  checkError_t(hipMalloc(&d_dep_size, sizeof(int)*num_nodes), "d_dep_size allocation failed");
  checkError_t(hipMalloc(&d_topo_result_gpu, sizeof(int)*num_nodes), "d_topo_result_gpu allocation failed");
  checkError_t(hipMalloc(&d_partition_result_gpu, sizeof(int)*num_nodes), "d_partition_result_gpu allocation failed");
  checkError_t(hipMalloc(&d_partition_counter_gpu, sizeof(int)*num_nodes), "d_partition_counter_gpu allocation failed");
  checkError_t(hipMalloc(&write_size, sizeof(uint32_t)), "write_size allocation failed");
  checkError_t(hipMalloc(&max_partition_id, sizeof(int)), "max_partition_id allocation failed");
  checkError_t(hipMalloc(&d_fu_partition, sizeof(int)*num_nodes), "d_fu_partition allocation failed");

  auto start = std::chrono::steady_clock::now();
  checkError_t(hipMemcpy(d_adjp, _adjp.data(), sizeof(int)*num_nodes, hipMemcpyHostToDevice), "d_adjp memcpy failed"); 
  checkError_t(hipMemcpy(d_adjncy, _adjncy.data(), sizeof(int)*num_edges, hipMemcpyHostToDevice), "d_adjncy memcpy failed"); 
  checkError_t(hipMemcpy(d_adjncy_size, _adjncy_size.data(), sizeof(int)*num_nodes, hipMemcpyHostToDevice), "d_adjncy_size memcpy failed"); 
  checkError_t(hipMemcpy(d_dep_size, _dep_size.data(), sizeof(int)*num_nodes, hipMemcpyHostToDevice), "d_dep_size memcpy failed"); 
  checkError_t(hipMemcpy(d_topo_result_gpu, source.data(), sizeof(int)*source.size(), hipMemcpyHostToDevice), "d_topo_result_gpu memcpy failed"); 
  checkError_t(hipMemcpy(d_partition_result_gpu, partition_result_gpu.data(), sizeof(int)*num_nodes, hipMemcpyHostToDevice), "d_partition_result_gpu memcpy failed"); 
  checkError_t(hipMemcpy(d_partition_counter_gpu, partition_counter_gpu.data(), sizeof(int)*num_nodes, hipMemcpyHostToDevice), "d_partition_counter_gpu memcpy failed"); 
  checkError_t(hipMemset(write_size, 0, sizeof(uint32_t)), "write_size memset failed");
  int max_partition_id_cpu = source.size() - 1;
  checkError_t(hipMemcpy(max_partition_id, &max_partition_id_cpu, sizeof(int), hipMemcpyHostToDevice), "max_partition_id memcpy failed"); 
  checkError_t(hipMemcpy(d_fu_partition, fu_partition.data(), sizeof(int)*num_nodes, hipMemcpyHostToDevice), "d_fu_partition memcpy failed"); 


  // invoke kernel
  unsigned num_block;
  while(read_size > 0) { 
    num_block = (read_size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    partition_gpu<<<num_block, BLOCK_SIZE>>>(
      d_adjp, d_adjncy, d_adjncy_size, d_dep_size,
      d_topo_result_gpu,
      d_partition_result_gpu,
      d_partition_counter_gpu,
      _partition_size,
      max_partition_id,
      read_offset, read_size, // [read_offset, read_offset + read_size - 1] are all the frontiers 
      write_size,
      d_fu_partition
    );

    // calculate where to read for next iteration
    read_offset += read_size;
    checkError_t(hipMemcpy(&read_size, write_size, sizeof(uint32_t), hipMemcpyDeviceToHost), "queue_size memcpy failed");

    // set write_size = 0 for next iteration 
    checkError_t(hipMemset(write_size, 0, sizeof(uint32_t)), "write_size rewrite failed");
  }

  checkError_t(hipMemcpy(partition_result_gpu.data(), d_partition_result_gpu, sizeof(int)*num_nodes, hipMemcpyDeviceToHost), "_partition_result_gpu memcpy failed"); 
  auto end = std::chrono::steady_clock::now();
  partition_time += std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
  checkError_t(hipMemcpy(&max_partition_id_cpu, max_partition_id, sizeof(int), hipMemcpyDeviceToHost), "max_partition_id_cpu memcpy failed"); 
 
  // assign partition IDs to nodes according to partition_result_gpu
  size_t index = 0;
  for(auto& node : _nodes) {
    node._cluster_id = partition_result_gpu[index]; 
    // if(partition_result_gpu[index] == -1) {
    //   std::cerr << "partition_result_gpu wrong...\n";
    //   std::exit(EXIT_FAILURE);
    // }
    ++index;
  }

  // also assign dep_cnt to nodes cuz it is needed in incremental partition
  for(auto& node : _nodes) {
    node._dep_cnt = node._fanins.size();
  } 

  // reset the _cluster_cnt
  _cluster_cnt.resize(max_partition_id_cpu+1, 0); // to avoid reallocation
  for(const auto& node : _nodes) {
    ++_cluster_cnt[node._cluster_id];
  }

  _max_cluster_id = max_partition_id_cpu;

  checkError_t(hipFree(d_adjp), "d_adjp free failed");
  checkError_t(hipFree(d_adjncy), "d_adjncy free failed");
  checkError_t(hipFree(d_adjncy_size), "d_adjncy_size free failed");
  checkError_t(hipFree(d_dep_size), "d_dep_size free failed");
  checkError_t(hipFree(d_topo_result_gpu), "d_topo_result_gpu free failed");
  checkError_t(hipFree(d_partition_result_gpu), "d_partition_result_gpu free failed");
  checkError_t(hipFree(d_partition_counter_gpu), "d_partition_counter_gpu free failed");
  checkError_t(hipFree(write_size), "write_size free failed");
  checkError_t(hipFree(max_partition_id), "max_partition_id free failed");
  checkError_t(hipFree(d_fu_partition), "fu_partition free failed");
 
}

}  // end of namespace ot. -----------------------------------------------------------------------



















